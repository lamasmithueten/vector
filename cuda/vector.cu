#include "hip/hip_runtime.h"
#include "header/config.h"
#include "header/csv.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAddition(int * A, int * B, int * result){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < SIZE)
	result[i] = A[i] + B[i];
}

__global__ void vectorSubtraction(int * A, int * B, int * result){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < SIZE)
	result[i] = A[i] - B[i];
}

__global__ void dotProduct(int * A, int * B, long * result ){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < SIZE)
	result[i] = A[i] * B[i];
}




int main(int argc, char **argv){
	if (argc != 3){
		fprintf(stderr, "Usage: %s <vector1.csv> <vector2.csv>\n", argv[0]);
		exit(EXIT_FAILURE);
	}

	int * vectorA = (int *) malloc(SIZE * sizeof(int));
	int * vectorB = (int *) malloc(SIZE * sizeof(int));
	int * result = (int *) malloc(SIZE * sizeof(int));
	int * d_vectorA, * d_vectorB, * d_result;
	long * dotProductPartialResult = (long *) malloc(SIZE*sizeof(long));
	long * d_dotProductPartialResult;
	long finalSum=0;
	int threadsPerBlock=16;
	int numBlocks= (SIZE + threadsPerBlock - 1) / threadsPerBlock;



	readVectorFromCSV(argv[1], vectorA);
	readVectorFromCSV(argv[2], vectorB);

	hipMalloc((void **)&d_vectorA, SIZE*sizeof(int));
	hipMalloc((void **)&d_vectorB, SIZE*sizeof(int));
	hipMalloc((void **)&d_result, SIZE*sizeof(int));
	hipMalloc((void **)&d_dotProductPartialResult,SIZE* sizeof(long));

	hipMemcpy(d_vectorA, vectorA, SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_vectorB, vectorB, SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_dotProductPartialResult, dotProductPartialResult,SIZE* sizeof(long),hipMemcpyHostToDevice);







for (int i = 0; i<1000; i++){
	vectorAddition<<<threadsPerBlock, numBlocks>>> (d_vectorA, d_vectorB, d_result);
	}
	hipMemcpy(result, d_result, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	writeVectorToCSV("result_addition_cuda.csv", result);

for (int i = 0; i<1000; i++){
	vectorSubtraction<<<threadsPerBlock, numBlocks>>>(d_vectorA, d_vectorB, d_result);
	}
	hipMemcpy(result, d_result, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	writeVectorToCSV("result_subtraction_cuda.csv", result);

for (int i = 0; i<1000; i++){
	dotProduct<<<threadsPerBlock, numBlocks>>>(d_vectorA, d_vectorB, d_dotProductPartialResult);
	}
	hipMemcpy(dotProductPartialResult, d_dotProductPartialResult,SIZE* sizeof(long), hipMemcpyDeviceToHost);
	for (int i = 0; i < SIZE; i++){
		finalSum += dotProductPartialResult[i];
	}
	writeScalarToFile("result_dot_product_cuda.txt", &finalSum);
	  //hipDeviceSynchronize();


	  hipFree(d_vectorA);
	  hipFree(d_vectorB);
	  hipFree(d_result);
	  hipFree(d_dotProductPartialResult);

	free(vectorA);
	free(vectorB);
	free(result);
	free(dotProductPartialResult);

	
	return EXIT_SUCCESS;
}
